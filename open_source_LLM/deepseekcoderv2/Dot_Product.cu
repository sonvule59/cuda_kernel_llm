#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    const int len = 32;
    float *a_device, *b_device, *result_device;
    float *a_host = new float[len];
    float *b_host = new float[len];
    float *c_host = new float[len];

    hipMalloc((void **)&a_device, len * sizeof(float));
    hipMalloc((void **)&b_device, len * sizeof(float));
    hipMalloc((void **)&result_device, len * sizeof(float));

    // Initialize host vectors a and b
    for (int i = 0; i < len; ++i) {
        a_host[i] = i + 1.f;
        b_host[i] = 2.f * i + 3.f;
    }

    hipMemcpy(a_device, a_host, len * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_device, b_host, len * sizeof(float), hipMemcpyHostToDevice);

    // Set up kernel launch configuration
    dim3 threadsPerBlock(32);
    dim3 blocksPerGrid((len + threadsPerBlock.x - 1) / threadsPerBlock.x, 1);

    dotProduct<<<blocksPerGrid, threadsPerBlock>>>(a_device, b_device, result_device, len);

    // Copy result back to the host
    hipMemcpy(c_host, result_device, len * sizeof(float), hipMemcpyDeviceToHost);

    printf("Dot product: \n");
    for (int i = 0; i < len; ++i) {
        printf("%f ", c_host[i]);
    }
    printf("\n");

    // Clean up resources
    hipFree(a_device);
    hipFree(b_device);
    hipFree(result_device);
    delete[] a_host;
    delete[] b_host;
    delete[] c_host;

    return 0;
}