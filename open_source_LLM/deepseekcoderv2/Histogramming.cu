#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hiprand/hiprand_kernel.h>

const int N = 1000000; // Input array size
const int numBins = 1024; // Number of bins

__global__ void kernel(int *input, int *output, int numBins) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        output[input[index]] += 1;
    }
}

int main(void) {
    int *input, *output;
    hiprandGenerator_t generator;
    hiprandState_t state;
    int *d_output;

    std::vector<int> h_input(N), h_output(numBins);
    hipMalloc((void **)&input, N * sizeof(int));
    hipMalloc((void **)&d_output, numBins * sizeof(int));

    hiprandCreateGenerator(&generator, CURAND_RNG_PSEUDO_RANDOM);
    curandInit(generator, CURAND_RNG_ALG_AES, CURAND_DETERMINISTIC);
    hiprandSetPseudoRandomGeneratorSeed(generator, 0);

    for (int i = 0; i < N; ++i) {
        h_input[i] = curandGenInteger(generator, 32767) + 1; // Generate random integers in range [1, 32768]
    }

    hipMemcpy(input, h_input.data(), N * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32, 1, 1);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x);
    kernel<<<numBlocks, threadsPerBlock>>>(input, d_output, numBins);

    hipMemcpy(h_output.data(), d_output, numBins * sizeof(int), hipMemcpyDeviceToHost);

    hiprandDestroyGenerator(generator);
    hipFree(input);
    hipFree(d_output);

    return 0;
}