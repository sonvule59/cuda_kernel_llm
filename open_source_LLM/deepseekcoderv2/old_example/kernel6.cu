// element_wise_mul.cu

#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

__global__ void elementWiseMul(const float* A, const float* B, float* C, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        C[index] = A[index] * B[index];
    }
}

int main() {
    const int N = 1 << 24; // Number of elements, e.g., 16777216 for 2^24
    size_t size = N * sizeof(float);

    // Host arrays
    float* h_A = new float[N];
    float* h_B = new float[N];
    float* h_C = new float[N];

    // Initialize host arrays A and B with some values
    for (int i = 0; i < N; ++i) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(2 * i); // Different from A for testing
    }

    // Device arrays
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    auto start = std::chrono::high_resolution_clock::now();
    elementWiseMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize(); // Ensure kernel is done before timing
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Kernel execution time: " << elapsed.count() << " seconds" << std::endl;

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Cleanup
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}