#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__device__ void lossFunction(float* z, float* y, float* dL, int c) {
    float L = 0.f;
    for (int j = 0; j < c; ++j) {
        L += logf(expf(z[j]) - expf(dL[j]));
    }
    L -= y[0] * dL[0];
    for (int j = 1; j < c; ++j) {
        L -= y[j] * (logf(expf(dL[j])) - logf(expf(z[j]) - expf(dL[j])));
    }
    __syncthreads();
    L /= c;
    if (threadIdx.x == 0) {
        dL[0] = L;
    }
}

void calculateCrossEntropyLoss(float* predictedLogits, float* trueLabels, float* loss, int N, int C) {
    float* dL = new float[C];

    dim3 blockSize(32);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, 1);

    float* dev_predictedLogits;
    float* dev_trueLabels;
    float* dev_dL;
    hipMalloc((void**)&dev_predictedLogits, N * C * sizeof(float));
    hipMalloc((void**)&dev_trueLabels, N * sizeof(float));
    hipMalloc((void**)&dev_dL, C * sizeof(float));

    hipMemcpy(dev_predictedLogits, predictedLogits, N * C * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_trueLabels, trueLabels, N * sizeof(float), hipMemcpyHostToDevice);

    lossFunction<<<gridSize, blockSize>>>(dev_predictedLogits, dev_trueLabels, dev_dL, C);
    hipMemcpy(loss, dev_dL, C * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(dev_predictedLogits);
    hipFree(dev_trueLabels);
    hipFree(dev_dL);
}
// Usage:
// Prepare input arrays
float predictedLogits[N][C];
float trueLabels[N];

// Call the function to compute the loss
float loss;
calculateCrossEntropyLoss(predictedLogits, trueLabels, &loss, N, C);