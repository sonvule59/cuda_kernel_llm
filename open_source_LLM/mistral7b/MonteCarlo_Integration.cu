#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <hiprand/hiprand_kernel.h>

__device__ double f(double x) {
    // Define your function here
    return x * x;
}

void integrateKernel(hiprandState* state, double* output, int N, double h, double a, double b) {
    double y = 0.0;
    double x = a + h * hiprand_uniform(state);
    for (int i = 0; i < N; ++i) {
        y += f(x);
        x += 2 * h;
    }
    __syncthreads();
    if (blockDim.x * blockIdx.x + threadIdx.x >= N) output[blockIdx.x] += y / N / h;
}

double integrate(int N, double a, double b) {
    int blocks = (b - a) * 1024 / (N * sizeof(double)) + 1;
    int threadsPerBlock = 1024;

    // Allocate device memory
    double* d_output;
    hipMalloc(&d_output, blocks * sizeof(double));

    hiprandGenerator_t generator;
    curandCreate(&generator);
    hiprandState state;
    hiprand_init(generator, 0, 0, &state);

    // Launch the kernel on the GPU
    integrateKernel<<<blocks, threadsPerBlock>>>(state, d_output, N, (b - a) / N, a, b);

    double result;
    hipMemcpy(&result, d_output, blocks * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_output);
    curandDestroy(generator);

    return result;
}