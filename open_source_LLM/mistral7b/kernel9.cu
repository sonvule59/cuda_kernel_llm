#include "hip/hip_runtime.h"
// **convolution_kernel.cu**

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void convolve(const float *input, const float *filter, float *output) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= N && idx < N + M) {
        float sum = 0.f;
        for (int i = max(0, idx - M + 1); i <= min(N, idx); ++i) {
            sum += input[i] * filter[idx - i];
        }
        output[blockIdx.x] = sum;
    }
}

// **main.cu**

#include <stdio.h>
#include "convolution_kernel.h"

void checkCudaErrors(hipError_t err, const char *name) {
    if (err != hipSuccess) {
        fprintf(stderr, "Fatal error: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main() {
    int N = 1024;
    int M = 32;

    float *input_d, *filter_d, *output_d, *output_h;
    float *padded_input_d;
    size_t bufferSize;

    checkCudaErrors(hipMalloc((void **)&input_d, N * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&filter_d, M * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&output_d, (N - M + 1) * sizeof(float)));
    checkCudaErrors(hipMallocManaged(&output_h, (N - M + 1) * sizeof(float)));
    bufferSize = N * sizeof(float);
    checkCudaErrors(hipMalloc((void **)&padded_input_d, bufferSize));

    // Zero-pad input signal and copy it to the device
    float paddingValue = 0.f;
    hiprandGenerator_t generator;
    hiprandCreateGenerator(&generator, CURAND_RNG_PSEUDO_RANDOM, NULL);
    checkCudaErrors(hiprandGenerateNormal(generator, input_d, N)); // Fill input with random values for this example
    hipMemcpy(padded_input_d, input_d, bufferSize, hipMemcpyDeviceToHost);
    for (int i = N; i < bufferSize; ++i) {
        padded_input_d[i] = paddingValue;
    }
    hipMemcpy(input_d, padded_input_d, bufferSize, hipMemcpyHostToDevice);

    // Copy filter to the device
    float hann[M] = {0.5f, 0.5f};
    for (int i = 2; i < M; ++i) {
        hann[i] = 0.5f * cos(2.0f * M_PI * (i - 1) / (M - 1)); // Hann window
    }
    hipMemcpy(filter_d, hann, M * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N - M + 1) / threadsPerBlock + ((N - M + 1) % threadsPerBlock != 0 ? 1 : 0);

    convolve<<<blocksPerGrid, threadsPerBlock>>>(input_d, filter_d, output_d);
    hipMemcpy(output_h, output_d, (N - M + 1) * sizeof(float), hipMemcpyDeviceToHost);

    // Output computation and printing
    float sum = 0.f;
    for (int i = 0; i < N - M + 1; ++i) {
        sum += output_h[i];
    }
    printf("Sum of the convolved signal: %f\n", sum);

    hipFree(input_d);
    hipFree(filter_d);
    hipFree(output_d);
    hipFree(padded_input_d);
    hipHostFree(output_h);
    hiprandDestroyGenerator(generator);

    return 0;
}